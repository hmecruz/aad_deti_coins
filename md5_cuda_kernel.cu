#include "hip/hip_runtime.h"
//
// Tomás Oliveira e Silva,  October 2024
//
// Arquiteturas de Alto Desempenho 2024/2025
//
// MD5 hash CUDA kernel code
//
// md5_cuda_kernel() --- each thread computes the MD5 hash of one message
//
// do not use this directy to search for DETI coins!
//

//
// needed stuff
//

typedef unsigned int u32_t;

#include "md5.h"

//
// the nvcc compiler stores x[] and state[] in registers (constant indices!)
//
// global thread number: n = threadIdx.x + blockDim.x * blockIdx.x
// global warp number: n >> 5
// warp thread number: n & 31
//

extern "C" __global__ __launch_bounds__(128,1) void cuda_md5_kernel(u32_t *interleaved32_data,u32_t *interleaved32_hash)
{
  u32_t n,a,b,c,d,state[4],x[16];

  //
  // get the global thread number
  //
  n = (u32_t)threadIdx.x + (u32_t)blockDim.x * (u32_t)blockIdx.x;
  //
  // adjust data and hash pointers
  //
  interleaved32_data = &interleaved32_data[(n >> 5u) * (32u * 13u) + (n & 31u)];
  interleaved32_hash = &interleaved32_hash[(n >> 5u) * (32u *  4u) + (n & 31u)];
  //
  // compute MD5 hash
  //
# define C(c)         (c)
# define ROTATE(x,n)  (((x) << (n)) | ((x) >> (32 - (n))))
# define DATA(idx)    interleaved32_data[32u * (idx)]
# define HASH(idx)    interleaved32_hash[32u * (idx)]
# define STATE(idx)   state[idx]
# define X(idx)       x[idx]
  CUSTOM_MD5_CODE();
# undef C
# undef ROTATE
# undef DATA
# undef HASH
# undef STATE
# undef X
}
