#include "hip/hip_runtime.h"
// Arquiteturas de Alto Desempenho 2024/2025
//
// DETI Coin CUDA Kernel Code
//
// deti_coins_cuda_kernel_search() --- each thread tries 95 potential DETI coins
//

typedef unsigned int u32_t;

#include "md5.h"
#include "search_utilities.h"

//
// the nvcc compiler stores x[] and state[] in registers (constant indices!)
//
// global thread number: n = threadIdx.x + blockDim.x * blockIdx.x
// global warp number: n >> 5
// warp thread number: n & 31
//

#define VAR1_IDX_AVX 10 // between 9 and 11
#define VAR2_IDX_AVX 11 // between 9 and 11


// DETI Coin Search CUDA Kernel
extern "C" __global__ __launch_bounds__(128, 1) void deti_coins_cuda_kernel_search(u32_t *storage_area, u32_t var1, u32_t var2){
    coin_t coin;
    u32_t n, a, b, c, d, state[4], x[16], hash[4];
    n = (u32_t)threadIdx.x + (u32_t)blockDim.x * (u32_t)blockIdx.x;

    initialize_deti_coin(&coin);
    coin.coin_as_ints[VAR1_IDX_AVX] = var1;
    coin.coin_as_ints[VAR2_IDX_AVX] = var2;

    coin.coin_as_ints[4] += (n % 64) << 0; n/= 64;
    coin.coin_as_ints[4] += (n % 64) << 8; n/= 64;
    coin.coin_as_ints[4] += (n % 64) << 16; n/= 64;
    coin.coin_as_ints[4] += (n % 64) << 24; n/= 64;

    for(n = 0; n < 95; n++){
        // compute MD5 hash
        #define C(c)         (c)
        #define ROTATE(x,n)  (((x) << (n)) | ((x) >> (32 - (n))))
        #define DATA(idx)    coin.coin_as_ints[idx]
        #define HASH(idx)    hash[idx]
        #define STATE(idx)   state[idx]
        #define X(idx)       x[idx]
            CUSTOM_MD5_CODE();
            if (hash[3] == 0){
                u32_t n = atomicAdd(storage_area, 13);
                if (n + 13 <= 1024){
                    for (int j = 0; j < 13; j++) {
                        storage_area[n + j] = coin.coin_as_ints[j];
                    }
                }
            }
        coin.coin_as_ints[12] += 1 << 16; // Try different combinations
    }
        #undef C
        #undef ROTATE
        #undef DATA
        #undef HASH
        #undef STATE
        #undef X
}



    


