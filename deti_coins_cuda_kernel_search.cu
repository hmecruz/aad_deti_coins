#include "hip/hip_runtime.h"
// Arquiteturas de Alto Desempenho 2024/2025
//
// DETI Coin CUDA Kernel Code
//
// deti_coins_cuda_kernel_search() --- each thread tries 95 potential DETI coins
//

typedef unsigned int u32_t;

#include "md5.h"
#include "search_utilities.h"

//
// the nvcc compiler stores x[] and state[] in registers (constant indices!)
//
// global thread number: n = threadIdx.x + blockDim.x * blockIdx.x
// global warp number: n >> 5
// warp thread number: n & 31
//

#define VAR1_IDX_AVX 10 // between 9 and 11
#define VAR2_IDX_AVX 11 // between 9 and 11

// CUDA-compatible initialize_deti_coin function
__device__ __host__ inline void initialize_deti_coin_cuda(coin_t *coin) {
    for (int i = 0; i < sizeof(coin->coin_as_chars); i++) {
        coin->coin_as_chars[i] = 0;
    }
    const char *prefix = "DETI coin ";
    for (int i = 0; i < 10; i++) {
        coin->coin_as_chars[i] = prefix[i];
    }
    for (int i = 10; i < 51; i++) {
        coin->coin_as_chars[i] = ' ';
    }
    coin->coin_as_chars[51] = '\n';
}

// DETI Coin Search CUDA Kernel
extern "C" __global__ __launch_bounds__(128, 1) void deti_coins_cuda_kernel_search(u32_t *storage_area, u32_t var1, u32_t var2){
    coin_t coin;
    u32_t n, a, b, c, d, state[4], x[16], hash[4];
    n = (u32_t)threadIdx.x + (u32_t)blockDim.x * (u32_t)blockIdx.x;

    initialize_deti_coin_cuda(&coin);
    coin.coin_as_ints[VAR1_IDX_AVX] = var1;
    coin.coin_as_ints[VAR2_IDX_AVX] = var2;

    coin.coin_as_ints[4] += (n % 64) << 0; n/= 64;
    coin.coin_as_ints[4] += (n % 64) << 8; n/= 64;
    coin.coin_as_ints[4] += (n % 64) << 16; n/= 64;
    coin.coin_as_ints[4] += (n % 64) << 24; n/= 64;

    for(n = 0; n < 95; n++){
        // compute MD5 hash
        #define C(c)         (c)
        #define ROTATE(x,n)  (((x) << (n)) | ((x) >> (32 - (n))))
        #define DATA(idx)    coin.coin_as_ints[idx]
        #define HASH(idx)    hash[idx]
        #define STATE(idx)   state[idx]
        #define X(idx)       x[idx]
            CUSTOM_MD5_CODE();
            if (hash[3] == 0){
                u32_t n = atomicAdd(storage_area, 13);
                if (n + 13 <= 1024){
                    for (int j = 0; j < 13; j++) {
                        storage_area[n + j] = coin.coin_as_ints[j];
                    }
                }
            }
        coin.coin_as_ints[12] += 1 << 16; // Try different combinations
    }
        #undef C
        #undef ROTATE
        #undef DATA
        #undef HASH
        #undef STATE
        #undef X
}



    


